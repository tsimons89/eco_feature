#include "hip/hip_runtime.h"
#include "Gpu_filter.cuh"


dim3 blocks;
dim3 threads;
Size Gpu_filter::size;
float* Gpu_filter::gpu_src;
float* Gpu_filter::gpu_dst;
bool Gpu_filter::initilized = false;


__global__ void filter_kernel( float *in, float *out,bool y_dir, bool subtract, int cols,int rows) {
	int x = blockIdx.x * blockDim.x + threadIdx.x; 
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int pixel_index = y*cols +x;
	if(x >= cols || y >= rows){
		//out[pixel_index] = 0;
		return;
	}

	float pixel = in[pixel_index];

	float adj_pixel,res;
	if(y_dir)
		adj_pixel = in[pixel_index + cols];
	else
		adj_pixel = in[pixel_index + 1];

	if(subtract)
		res = adj_pixel - pixel;
	else
		res = adj_pixel + pixel;

	out[pixel_index] = res/2;
}

Mat Gpu_filter::apply(const vector<float*> gpu_images){
	Mat ret_images;
	for(int i = 0; i < gpu_images.size();i++)
		ret_images.push_back(apply(gpu_images.at(i)));
	return ret_images;
}

Mat Gpu_filter::apply(float* gpu_image){
	this->gpu_image = gpu_image;
	first = true;
	blur_x(genome.x_blur);
	blur_y(genome.y_blur);
	diff_x(genome.x_diff);
	diff_y(genome.y_diff);
	return download();	
}

float* Gpu_filter::upload(Mat image){
	if(!initilized)
		init(image.size());
	float* image_data = (float*)image.data;
	float* ret_image;
	hipMalloc((void**)&ret_image, size.area() * sizeof(float));
	hipMemcpy( ret_image, image_data, size.area() * sizeof(float),hipMemcpyHostToDevice);
	return ret_image;
}

void Gpu_filter::init(Size s){
	size = s;
	hipMalloc((void**)&gpu_src, size.area() * sizeof(float));
	hipMalloc((void**)&gpu_dst, size.area() * sizeof(float));
	initilized = true;
	blocks = dim3((size.width + THREAD_DIM - 1)/THREAD_DIM ,(size.height + THREAD_DIM - 1)/THREAD_DIM);
	threads = dim3(THREAD_DIM,THREAD_DIM);
}

Mat Gpu_filter::download(){
	float* data = (float*)malloc(size.area()*sizeof(float));
	hipMemcpy(data,gpu_src,size.area()*sizeof(float),hipMemcpyDeviceToHost);
	Mat res_image(size,CV_32F,data);
	Size new_size(size.width - (genome.x_blur+genome.x_diff),size.height - (genome.y_blur + genome.y_diff));
	Rect roi(Point(0,0),new_size);
	Mat small_image(res_image,roi);
	return Mat(res_image,roi);
}

void Gpu_filter::filter(bool y_dir, bool subtract){
	if(first){
		first = false;
		filter_kernel<<<blocks,threads>>>(gpu_image,gpu_dst,y_dir,subtract,size.width,size.height);
	}
	else
		filter_kernel<<<blocks,threads>>>(gpu_src,gpu_dst,y_dir,subtract,size.width,size.height);
	float * tmp = gpu_src;
	gpu_src = gpu_dst;
	gpu_dst = tmp;
}

void Gpu_filter::blur_x(int n){
	for(int i = 0; i < n; i++)
		filter(false,false);
}

void Gpu_filter::blur_y(int n){
	for(int i = 0; i < n; i++)
		filter(true,false);
}

void Gpu_filter::diff_x(int n){
	for(int i = 0; i < n; i++)
		filter(false,true);
}

void Gpu_filter::diff_y(int n){
	for(int i = 0; i < n; i++)
		filter(true,true);
}



